#include "hip/hip_runtime.h"
#include <iostream>
#include <vector>
#include <cstdlib>
#include <cstdio>
#include <unistd.h>

#include "gtest/gtest.h"
#include "utils.h"

using std::vector;

int n = 1024;
int n_thread = 512;

__global__ void Initialize(int n, int *a, int *b)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if (i < n && j < n)
    {
        a[n * i + j] = j;
        b[n * i + j] = i - 2 * j;
    }
}

__global__ void Add(int n, int *a, int *b, int *c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    if (i < n && j < n)
    {
        c[n * i + j] = a[n * i + j] + b[n * i + j];
    }
}

TEST(CUDA, add_matrices)
{
    printf("Dimensions of matrix: %5d x %5d\n", n, n);

    int *d_a, *d_b, *d_c;

    /* Allocate memory */
    checkCudaErrors(hipMalloc(&d_a, sizeof(int) * n * n));
    checkCudaErrors(hipMalloc(&d_b, sizeof(int) * n * n));
    checkCudaErrors(hipMalloc(&d_c, sizeof(int) * n * n));

    ASSERT_GT(n_thread, 0) << "The number of threads should be a positive number";
    ASSERT_EQ(n_thread % 32, 0) << "The number of threads should be a multiple of 32";
    ASSERT_LE(n_thread, 1024) << "The number of threads should be smaller than 1024";

    dim3 th_block(32, n_thread / 32);

    ASSERT_LE(th_block.x * th_block.y, 1024);
    ASSERT_EQ(th_block.x, 32);
    ASSERT_GT(th_block.y, 0);

    int blocks_per_grid_x = (n + th_block.x - 1) / th_block.x;
    int blocks_per_grid_y = (n + th_block.y - 1) / th_block.y;
    /* This formula is needed to make sure we process all entries in matrix */
    dim3 num_blocks(blocks_per_grid_x, blocks_per_grid_y);

    printf("Dimension of thread block: %2d x %2d\n", th_block.x, th_block.y);
    printf("Dimension of grid: %3d x %3d\n", num_blocks.x, num_blocks.y);

    /* Run calculation on GPU */
    Initialize<<<num_blocks, th_block>>>(n, d_a, d_b);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    Add<<<num_blocks, th_block>>>(n, d_a, d_b, d_c);
    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    /* Note that kernels execute asynchronously.
       They will fail without any error message!
       This can be confusing when debugging.
       The output arrays will be left uninitialized with no warning.
       */

    vector<int> h_c(n * n);
    /* Copy the result back */
    checkCudaErrors(hipMemcpy(&h_c[0], d_c, sizeof(int) * n * n,
                               hipMemcpyDeviceToHost));

    /* Test result */
    for (int i = 0; i < n; ++i)
    {
        for (int j = 0; j < n; ++j)
        {
            ASSERT_EQ(h_c[n * i + j], i - j);
        }
    }
}

int main(int argc, char **argv)
{
    if (checkCmdLineFlag(argc, argv, "t"))
    {
        n_thread = getCmdLineArgumentInt(argc, argv, "t");
        printf("Using %d threads = %d warps\n", n_thread, (n_thread + 31) / 32);
    }

    ::testing::InitGoogleTest(&argc, argv);
    return RUN_ALL_TESTS();
}
