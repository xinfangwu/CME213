#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>

#include "BC.h"

/**
 * Calculates the next finite difference step given a
 * grid point and step lengths.
 *
 * @param curr Pointer to the grid point that should be updated.
 * @param width Number of grid points in the x dimension.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 * @returns Grid value of next timestep.
 */
template <int order>
__device__ float Stencil(const float *curr, int width, float xcfl, float ycfl)
{
    switch (order)
    {
    case 2:
        return curr[0] + xcfl * (curr[-1] + curr[1] - 2.f * curr[0]) +
               ycfl * (curr[width] + curr[-width] - 2.f * curr[0]);

    case 4:
        return curr[0] + xcfl * (-curr[2] + 16.f * curr[1] - 30.f * curr[0] + 16.f * curr[-1] - curr[-2]) + ycfl * (-curr[2 * width] + 16.f * curr[width] - 30.f * curr[0] + 16.f * curr[-width] - curr[-2 * width]);

    case 8:
        return curr[0] + xcfl * (-9.f * curr[4] + 128.f * curr[3] - 1008.f * curr[2] + 8064.f * curr[1] - 14350.f * curr[0] + 8064.f * curr[-1] - 1008.f * curr[-2] + 128.f * curr[-3] - 9.f * curr[-4]) + ycfl * (-9.f * curr[4 * width] + 128.f * curr[3 * width] - 1008.f * curr[2 * width] + 8064.f * curr[width] - 14350.f * curr[0] + 8064.f * curr[-width] - 1008.f * curr[-2 * width] + 128.f * curr[-3 * width] - 9.f * curr[-4 * width]);

    default:
        printf("ERROR: Order %d not supported", order);
        return HIP_NAN_F;
    }
}

/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be very simple and only use global memory
 * and 1d threads and blocks.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param nx Number of grid points in the x dimension to which the full
 *           stencil can be applied (ie the number of points that are at least
 *           order/2 grid points away from the boundary).
 * @param ny Number of grid points in the y dimension to which th full
 *           stencil can be applied.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template <int order>
__global__ void gpuStencilGlobal(float *next, const float *__restrict__ curr, int gx, int nx, int ny,
                                 float xcfl, float ycfl)
{
    const int borderSize = order / 2;
    const int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid >= nx * ny)
        return;

    const int x = tid % nx + borderSize;
    const int y = tid / nx + borderSize;

    const int gl = y * gx + x;
    next[gl] = Stencil<order>(&curr[gl], gx, xcfl, ycfl);
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuStencilGlobal kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
double gpuComputationGlobal(Grid &curr_grid, const simParams &params)
{

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    const int gx = params.gx();
    const int nx = params.nx();
    const int ny = params.ny();

    const int N = nx * ny;

    const int num_threads = 1024;
    const int num_blocks = (N + num_threads - 1) / num_threads;

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i)
    {
        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // apply stencil
        switch (params.order())
        {
        case 2:
            gpuStencilGlobal<2><<<num_blocks, num_threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;

        case 4:
            gpuStencilGlobal<4><<<num_blocks, num_threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;

        case 8:
            gpuStencilGlobal<8><<<num_blocks, num_threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;
        }

        Grid::swap(curr_grid, next_grid);
    }

    check_launch("gpuStencilGlobal");
    return stop_timer(&timer);
}

/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be optimized to compute finite difference updates
 * in blocks of size (blockDim.y * numYPerStep) * blockDim.x. Each thread
 * should calculate at most numYPerStep updates. It should still only use
 * global memory.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param nx Number of grid points in the x dimension to which the full
 *           stencil can be applied (ie the number of points that are at least
 *           order/2 grid points away from the boundary).
 * @param ny Number of grid points in the y dimension to which th full
 *           stencil can be applied.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template <int order, int numYPerStep>
__global__ void gpuStencilBlock(float *next, const float *__restrict__ curr, int gx, int nx, int ny,
                                float xcfl, float ycfl)
{
    const unsigned borderSize = order >> 1;
    const unsigned tidx = blockIdx.x * blockDim.x + threadIdx.x + borderSize;

    if (tidx >= nx + borderSize)
    {
        return;
    }

    const unsigned tidy = blockIdx.y * blockDim.y * numYPerStep + threadIdx.y +
                          borderSize;
    const size_t g_start = gx * tidy + tidx;
    const unsigned g_stride = gx * blockDim.y;

    const unsigned g_end1 = tidy + blockDim.y * numYPerStep;
    // end of y loop if we compute numYPerStep iterations
    const unsigned g_end2 = ny + borderSize;
    // end of grid along y dimension

    if (g_end1 <= g_end2)
    {
        // We can safely perform numYPerStep iterations
        unsigned gl = g_start;

        for (unsigned i = 0; i < numYPerStep; ++i)
        {
            next[gl] = Stencil<order>(&curr[gl], gx, xcfl, ycfl);
            gl += g_stride;
        }
    }
    else
    {
        const size_t g_end = gx * (ny + borderSize) + tidx;
        // The thread stops once it reaches the end of the grid

        for (unsigned gl = g_start; gl < g_end; gl += g_stride)
        {
            next[gl] = Stencil<order>(&curr[gl], gx, xcfl, ycfl);
        }
    }
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuStencilBlock kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
double gpuComputationBlock(Grid &curr_grid, const simParams &params)
{

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    const int numThreadx = 64; /* opt 1: 64; opt 2: 32 */
    const int numThready = 8;
    const int numYPerStep = 8; /* opt 1: 8; opt 2: 4 */

    const int gx = params.gx();
    const int nx = params.nx();
    const int ny = params.ny();

    dim3 threads(numThreadx, numThready);
    dim3 blocks((nx + numThreadx - 1) / numThreadx,
                (ny + numThready * numYPerStep - 1) /
                    (numYPerStep * numThready));

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i)
    {
        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // apply stencil
        switch (params.order())
        {
        case 2:
            gpuStencilBlock<2, numYPerStep><<<blocks, threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;

        case 4:
            gpuStencilBlock<4, numYPerStep><<<blocks, threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;

        case 8:
            gpuStencilBlock<8, numYPerStep><<<blocks, threads>>>(
                next_grid.dGrid_, curr_grid.dGrid_, gx, nx, ny,
                params.xcfl(), params.ycfl());
            break;
        }
        Grid::swap(curr_grid, next_grid);
    }

    check_launch("gpuStencilBlock");
    return stop_timer(&timer);
}

/**
 * Kernel to propagate finite difference grid from the current
 * time point to the next.
 *
 * This kernel should be optimized to compute finite difference updates
 * in blocks of size side * side using shared memory.
 *
 * @param next[out] Next grid state.
 * @param curr Current grid state.
 * @param gx Number of grid points in the x dimension.
 * @param gy Number of grid points in the y dimension.
 * @param xcfl Courant number for x dimension.
 * @param ycfl Courant number for y dimension.
 */
template <int side, int order>
__global__ void gpuStencilShared(float *next, const float *__restrict__ curr, int gx, int gy,
                                 float xcfl, float ycfl)
{
    const int borderSize = order / 2;
    const int usefulSide = side - order;
    const int xpos = threadIdx.x;
    const int ylane = threadIdx.y; // See the definition of ypos below
    const int globalXPos = blockIdx.x * usefulSide + xpos;
    const int numYPerStep = side / blockDim.y;
    const int localStride = blockDim.y * side;
    const int globalStride = blockDim.y * gx;

    __shared__ float smem[side * side];

    // Use threads to load our slice into smem
    if (globalXPos < gx)
    {
        int localOffset = ylane * side + xpos;
        int globalOffset = (blockIdx.y * usefulSide + ylane) * gx +
                           globalXPos;

        if (side <= gy - blockIdx.y * usefulSide)
        {
            // This block of threads will perform a full
            // side x side update with numYPerStep iterations
            for (int i = 0; i < numYPerStep; ++i)
            {
                smem[localOffset] = curr[globalOffset];
                localOffset += localStride;
                globalOffset += globalStride;
            }
        }
        else
        {
            const int ypos_end = gy - blockIdx.y * usefulSide;
            // End of y iterations for this block
            const int local_end = ypos_end * side + xpos;
            // End of loop using localOffset

            for (; localOffset < local_end; localOffset += localStride)
            {
                smem[localOffset] = curr[globalOffset];
                globalOffset += globalStride;
            }
        }

        // Code variant which does not use a separate loop with numYPerStep
        // performance is lower

        // int localOffset = ylane * side + xpos;
        // int globalOffset = (blockIdx.y * usefulSide + ylane) * gx +
        //                    globalXPos;

        // const int ypos_end = min(side, gy - blockIdx.y * usefulSide);
        // // End of y iterations for this block
        // const int local_end = ypos_end * side + xpos;
        // // End of loop using localOffset

        // for(; localOffset < local_end; localOffset += localStride) {
        //     smem[localOffset] = curr[globalOffset];
        //     globalOffset += globalStride;
        // }
    }

    __syncthreads();

    // Now that everything is loaded in smem, do the stencil calculation.
    if (globalXPos < gx - borderSize && xpos >= borderSize &&
        xpos < side - borderSize)
    {

        int localOffset = (ylane + borderSize) * side + xpos;
        int globalOffset = (blockIdx.y * usefulSide + ylane + borderSize) * gx + globalXPos;

        const int ypos_end = min(side - borderSize,
                                 gy - borderSize - blockIdx.y * usefulSide);
        const int local_end = ypos_end * side + xpos;

        for (; localOffset < local_end; localOffset += localStride)
        {
            next[globalOffset] =
                Stencil<order>(&smem[localOffset], side, xcfl, ycfl);
            globalOffset += globalStride;
        }

        // Code variant with a loop over a separate index i
        // performand is lower

        // const int i_end = (local_end - localOffset + localStride-1) / localStride;
        // for(int i = 0; i<i_end; ++i) {
        //     next[globalOffset] =
        //         Stencil<order>(&smem[localOffset], side, xcfl, ycfl);
        //     localOffset += localStride;
        //     globalOffset += globalStride;
        // }
    }
}

/**
 * Propagates the finite difference 2D heat diffusion solver
 * using the gpuShared kernel.
 *
 * Use this function to do necessary setup and propagate params.iters()
 * number of times.
 *
 * @param curr_grid The current state of the grid.
 * @param params Parameters for the finite difference computation.
 * @returns Time required for computation.
 */
template <int order>
double gpuComputationShared(Grid &curr_grid, const simParams &params)
{

    boundary_conditions BC(params);

    Grid next_grid(curr_grid);

    const int numThreadx = 64; // 64;
    const int numThready = 2;  // 8;

    const int gx = params.gx();
    const int gy = params.gy();
    const int nx = params.nx();
    const int ny = params.ny();

    const int smemSide = numThreadx;
    const int usefulsmemSide = smemSide - order;

    const int numBlocksX = (nx + usefulsmemSide - 1) / usefulsmemSide;
    const int numBlocksY = (ny + usefulsmemSide - 1) / usefulsmemSide;

    dim3 threads(numThreadx, numThready);
    dim3 blocks(numBlocksX, numBlocksY);

    event_pair timer;
    start_timer(&timer);

    for (int i = 0; i < params.iters(); ++i)
    {
        // update the values on the boundary only
        BC.updateBC(next_grid.dGrid_, curr_grid.dGrid_);

        // apply stencil
        gpuStencilShared<smemSide, order><<<blocks, threads>>>(
            next_grid.dGrid_, curr_grid.dGrid_, gx, gy,
            params.xcfl(), params.ycfl());

        Grid::swap(curr_grid, next_grid);
    }

    check_launch("gpuStencilShared");
    return stop_timer(&timer);
}
