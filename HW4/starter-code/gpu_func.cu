#include "hip/hip_runtime.h"
#define ARMA_ALLOW_FAKE_GCC
#include <algorithm>
#include <armadillo>
#include <cassert>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <helper_functions.h>
#include <iostream>
#include <memory>

#include "gpu_func.h"
#include "util.cuh"

__global__ void Warmup() {}

void DWarmup() { Warmup<<<1, 1>>>(); }

/**
 * DeviceAllocator and DeviceMatrix
 */

DeviceAllocator::DeviceAllocator(nn_real *cpu_data, int n) {
  assert(n >= 0);
  assert(cpu_data != nullptr);
  nbytes = n * sizeof(nn_real);
  hipMalloc(&data, nbytes);
  hipMemcpy(data, cpu_data, nbytes, hipMemcpyHostToDevice);
}

DeviceAllocator::DeviceAllocator(int n) {
  assert(n >= 0);
  nbytes = n * sizeof(nn_real);
  hipMalloc(&data, nbytes);
}

DeviceAllocator::~DeviceAllocator() {
  if (data != nullptr)
    hipFree(data);
}

int DeviceAllocator::total_bytes() { return nbytes; }

nn_real *DeviceAllocator::memptr() { return data; }

void DeviceAllocator::to_cpu(nn_real *cpu_data) {
  assert(data != nullptr && cpu_data != nullptr);
  hipMemcpy(cpu_data, data, nbytes, hipMemcpyDeviceToHost);
}

DeviceMatrix::DeviceMatrix(int n_rows, int n_cols) {
  assert(n_rows >= 0 && n_cols >= 0);
  this->allocator = std::make_shared<DeviceAllocator>(n_rows * n_cols);
  this->data = this->allocator->memptr();
  this->n_rows = n_rows;
  this->n_cols = n_cols;
}

DeviceMatrix::DeviceMatrix(arma::Mat<nn_real> &cpu_mat) {
  this->allocator = std::make_shared<DeviceAllocator>(
      cpu_mat.memptr(), cpu_mat.n_rows * cpu_mat.n_cols);
  this->data = this->allocator->memptr();
  this->n_rows = cpu_mat.n_rows;
  this->n_cols = cpu_mat.n_cols;
}

int DeviceMatrix::total_bytes() { return allocator->total_bytes(); }

nn_real *DeviceMatrix::memptr() { return data; }

void DeviceMatrix::to_cpu(arma::Mat<nn_real> &cpu_mat) {
  allocator->to_cpu(cpu_mat.memptr());
}

__device__ nn_real &DeviceMatrix::operator()(int row, int col, bool transpose) {
  assert(data != nullptr && row >= 0 && row < n_rows && col >= 0 &&
         col < n_cols);
  return transpose ? data[row * n_cols + col] : data[col * n_rows + row];
}
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //
//                           GEMM kernels                           //
// +-*=+-*=+-*=+-*=+-*=+-*=+-*=+-*=+*-=+-*=+*-=+-*=+-*=+-*=+-*=+-*= //

__global__ void BasicMatMulColumnMajor(DeviceMatrix A, DeviceMatrix B,
                                       DeviceMatrix C, nn_real alpha,
                                       nn_real beta) {
  // V TODO: Implement this kernel
  int row = blockDim.x * blockIdx.x + threadIdx.x;
  int col = blockDim.y * blockIdx.y + threadIdx.y;
  

  if(col < C.n_cols && row < C.n_rows){
    nn_real sum = 0;
    for(int k =0; k< A.n_cols; k++){
      sum += A(row, k) * B(k, col);
    }
    C(row, col) = alpha * sum + beta * C(row, col);
  }
}

void basicGEMMColumnMajor(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                          nn_real alpha, nn_real beta) {
  // V TODO: Implement this kernel wrapper
  // Remember that column major means that consecutive threads compute
  // consecutive elements in a column of the output matrix

  // check_launch("basicGEMMColumnMajor");
  int numThread_row = 32;
  int numThread_col = 32;
  int numBlock_row = (C.n_rows + numThread_row - 1)/numThread_row;
  int numBlock_col = (C.n_cols + numThread_col - 1)/numThread_col;
  dim3 blockSize(numThread_row, numThread_col);
  dim3 gridSize(numBlock_row, numBlock_col);

  // Launch the kernel
  BasicMatMulColumnMajor<<<gridSize, blockSize>>>(A, B, C, alpha, beta);

  // Check for errors in kernel launch or during execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}

__global__ void BasicMatMulRowMajor(DeviceMatrix A, DeviceMatrix B,
                                    DeviceMatrix C, nn_real alpha,
                                    nn_real beta) {
  // V TODO: Implement this kernel
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  
  if(col < C.n_cols && row < C.n_rows){
    nn_real sum = 0;
    for(int k =0; k< A.n_cols; k++){
      sum += A(row, k) * B(k, col);
    }
    C(row, col) = alpha * sum + beta * C(row, col);
  }

}

void basicGEMMRowMajor(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                       nn_real alpha, nn_real beta) {
  // V TODO: Implement this kernel wrapper
  // Remember that row major means that consecutive threads compute
  // consecutive elements in a row of the output matrix

  // check_launch("basicGEMMRowMajor");
  int numThread_row = 128;
  int numThread_col = 8;
  int numBlock_row = (C.n_rows + numThread_row - 1)/numThread_row;
  int numBlock_col = (C.n_cols + numThread_col - 1)/numThread_col;
  dim3 blockSize(numThread_col, numThread_row);
  dim3 gridSize(numBlock_col, numBlock_row);

  // Launch the kernel
  BasicMatMulRowMajor<<<gridSize, blockSize>>>(A, B, C, alpha, beta);

  // Check for errors in kernel launch or during execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
  }

}

#define blockSize_x 32
#define blockSize_y 32

template <int blockSizeX, int blockSizeY>
__global__ void SharedMemoryMatMul(DeviceMatrix A, DeviceMatrix B,
                                   DeviceMatrix C, nn_real alpha,
                                   nn_real beta) {

  // V TODO: Implement this kernel
  // x-> col, y -> row
  int row = blockDim.y * blockIdx.y + threadIdx.y;
  int col = blockDim.x * blockIdx.x + threadIdx.x;
  __shared__ nn_real As[blockSizeX][blockSizeY];
  __shared__ nn_real Bs[blockSizeX][blockSizeY];
  nn_real Cvalue = 0;

  // loop over all sub-matrices of A and B along ->
  for(int m=0; m<((A.n_cols + blockSizeX - 1)/blockSizeX); m++){
    
    // load to shared As
    // one thread one element 
    int in_col = m * blockSizeX + threadIdx.x;
    if(in_col < A.n_cols && row < A.n_rows){
      As[threadIdx.y][threadIdx.x] = A(row, in_col);
    }
    else{
      As[threadIdx.y][threadIdx.x] = 0;
    }

    // load to shared Bs
    // one thread one element 
    int in_row = m * blockSizeY + threadIdx.y;
    if(in_row < B.n_rows && col < B.n_cols){
      Bs[threadIdx.y][threadIdx.x] = B(in_row, col);
    }
    else{
      Bs[threadIdx.y][threadIdx.x] = 0;
    }

    __syncthreads();

    // do sum
    for (int e=0; e<blockSizeY; e++){
      Cvalue += As[threadIdx.y][e] * Bs[e][threadIdx.x];
    }
    __syncthreads();
  }

  if (row < C.n_rows && col < C.n_cols){
    C(row, col) = alpha * Cvalue + beta * C(row, col);
  }
}

void sharedMemoryGEMM(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C,
                      nn_real alpha, nn_real beta) {
  // V TODO: Implement this wrapper
  // check_launch("sharedMemoryGEMM");
  // x -> col, y -> row
  int numThread_x = blockSize_x;
  int numThread_y = blockSize_y;
  int numBlock_x = (C.n_cols + numThread_x - 1)/numThread_x;
  int numBlock_y = (C.n_rows + numThread_y - 1)/numThread_y;
  dim3 blockSize(numThread_x, numThread_y);
  dim3 gridSize(numBlock_x, numBlock_y);

  // Launch the kernel
  SharedMemoryMatMul<blockSize_x, blockSize_y><<<gridSize, blockSize>>>(A, B, C, alpha, beta);

  // Check for errors in kernel launch or during execution
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
      printf("CUDA error: %s\n", hipGetErrorString(err));
  }
}

// 32x32 Hierarchical Tiling
// num_thread: number of threads per block
// blockItemsM: number of rows of A in each submatrix of A
// blockItemsN: number of columns of B in each submatrix of B
// blockItemsK: number of columns in submatrix of A and rows in submatrix of B
template <int num_thread, int blockItemsM, int blockItemsN, int blockItemsK>
__global__ void TiledMatMul(DeviceMatrix A, bool transa, DeviceMatrix B,
                            bool transb, DeviceMatrix C, nn_real alpha,
                            nn_real beta) {
  // TODO: Implement this kernel
  // x-> col, y -> row
  // int row = blockDim.y * blockIdx.y + threadIdx.y;
  // int col = blockDim.x * blockIdx.x + threadIdx.x;
  // __shared__ nn_real As[blockItemsM][blockItemsK];
  // __shared__ nn_real Bs[blockItemsK][blockItemsN];
  // nn_real Cvalue = 0;

  // int K = A.n_cols;

  // // First, Load A tile and B tile into shared memory 
  // // Matrix -> Tile
  // for(int kblock=0; kblock<((K + blockItemsK -1)/blockItemsK); kblock++){

  //   int in_col = kblock * blockItemsK + threadIdx.x;
  //   if(in_col < A.n_cols && row < A.n_rows){
  //     As[threadIdx.y][threadIdx.x] = A(row, in_col);
  //   }
  //   else{
  //     As[threadIdx.y][threadIdx.x] = 0;
  //   }

  //   // load to shared Bs
  //   // one thread one element 
  //   int in_row = kblock * blockItemsK + threadIdx.y;
  //   if(in_row < B.n_rows && col < B.n_cols){
  //     Bs[threadIdx.y][threadIdx.x] = B(in_row, col);
  //   }
  //   else{
  //     Bs[threadIdx.y][threadIdx.x] = 0;
  //   }

  //   __syncthreads();

  //   // Second, Tile -> Warp 
  //   int WarpSizeY = 16;
  //   int WarpSizeX = 8;
  //   int warpRow = threadIdx.y / WarpSizeY;
  //   int warpCol = threadIdx.x / WarpSizeX;
  //   // int laneIndex = threadIdx.y % WarpSize * WarpSize + threadIdx.x % WarpSize;

  //   #pragma unroll
  //   for (int k = 0; k < blockItemsK; k++) {
  //       nn_real frag_a = As[warpRow * WarpSize + laneIndex / ThreadItemsX][k];
  //       nn_real frag_b = Bs[k][warpCol * WarpSize + laneIndex % ThreadItemsX];
  //       #pragma unroll
  //       for (int thread_x = 0; thread_x < ThreadItemsX; ++thread_x) {
  //           #pragma unroll
  //           for (int thread_y = 0; thread_y < ThreadItemsY; ++thread_y) {
  //               Cvalue += frag_a[y] * frag_b[x];
  //           }
  //       }
  //   }
  //   __syncthreads();

  // }

  // if (row < C.n_rows && col < C.n_cols){
  //   C(row, col) = alpha * Cvalue + beta * C(row, col);
  // }
  // check_launch("tiledGEMM");
}


// wrapper for MatMulTile_32_32
void tiledGEMM(DeviceMatrix A, DeviceMatrix B, DeviceMatrix C, nn_real alpha,
               nn_real beta) {
  assert((A.n_cols) == (B.n_rows));
  assert(C.n_rows == (A.n_rows) && C.n_cols == (B.n_cols));

  constexpr int block_m = 32;
  constexpr int block_n = 32;
  constexpr int block_k = 32;
  constexpr int num_thread = 128;
  dim3 grid((C.n_rows + block_m - 1) / block_m,
            (C.n_cols + block_n - 1) / block_n);
  TiledMatMul<num_thread, block_m, block_n, block_k>
      <<<grid, num_thread>>>(A, false, B, false, C, alpha, beta);

  check_launch("tiledGEMM");
}
